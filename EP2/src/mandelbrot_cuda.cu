#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include<hip/hip_runtime.h>
#define SIZE 10
#define iteration_max 200
#define gradient_size 16

struct timer_info {
    clock_t c_start;
    clock_t c_end;
    struct timespec t_start;
    struct timespec t_end;
    struct timeval v_start;
    struct timeval v_end;
};

struct timer_info timer, timerAloc;

typedef struct param{
    double c_x_min;
    double c_x_max;
    double c_y_min;
    double c_y_max;
    double pixel_width;
    double pixel_height;
    int image_size;
    int i_x_max;
    int i_y_max;
    int image_buffer_size;
    unsigned char *image_buffer;
    int *colors;
} params;

typedef params* Params;

void allocate_image_buffer(Params p){
    int rgb_size = 3;
    hipMallocManaged(&(p->image_buffer), sizeof(unsigned char) * p->image_buffer_size * rgb_size);
};

Params CreateParams(){
    Params p;
    hipMallocManaged(&p,sizeof(Params));
    hipMallocManaged(&(p->colors),51*sizeof(int));
    allocate_image_buffer(p);
    int colorvals[51]={
    66, 30, 15,
    25, 7, 26,
    9, 1, 47,
    4, 4, 73,
    0, 7, 100,
    12, 44, 138,
    24, 82, 177,
    57, 125, 209,
    134, 181, 229,
    211, 236, 248,
    241, 233, 191,
    248, 201, 95,
    255, 170, 0,
    204, 128, 0,
    153, 87, 0,
    106, 52, 3,
    16, 16, 16,
    };
    for(int i=0;i<51;i++){
        p->colors[i]=colorvals[i];
    }
    return p;
}


void init(int argc, char *argv[],Params p){
    if(argc < 6){
        printf("usage: ./mandelbrot_seq c_x_min c_x_max c_y_min c_y_max image_size\n");
        printf("examples with image_size = 11500:\n");
        printf("    Full Picture:         ./mandelbrot_seq -2.5 1.5 -2.0 2.0 11500\n");
        printf("    Seahorse Valley:      ./mandelbrot_seq -0.8 -0.7 0.05 0.15 11500\n");
        printf("    Elephant Valley:      ./mandelbrot_seq 0.175 0.375 -0.1 0.1 11500\n");
        printf("    Triple Spiral Valley: ./mandelbrot_seq -0.188 -0.012 0.554 0.754 11500\n");
        exit(0);
    } 
    else{
        printf("AAAA\n");
        sscanf(argv[1], "%lf", &(p->c_x_min));
        sscanf(argv[2], "%lf", &(p->c_x_max));
        sscanf(argv[3], "%lf", &(p->c_y_min));
        sscanf(argv[4], "%lf", &(p->c_y_max));
        sscanf(argv[5], "%d", &(p->image_size));
        p->i_x_max           = p->image_size;
        p->i_y_max           = p->image_size;
        p->image_buffer_size = p->image_size * p->image_size;
    
        p->pixel_width       = (double)(p->c_x_max - p->c_x_min) / p->i_x_max;
        p->pixel_height      = (double)(p->c_y_max - p->c_y_min) / p->i_y_max;
    };
};
/*
void update_rgb_buffer(int iteration, int x, int y,Params p){
    int color;

    if(iteration == iteration_max){
        image_buffer[(p.i_y_max * y) + x][0] = colors[gradient_size][0];
        image_buffer[(p.i_y_max * y) + x][1] = colors[gradient_size][1];
        image_buffer[(p.i_y_max * y) + x][2] = colors[gradient_size][2];
    }
    else{
        color = iteration % gradient_size;

        image_buffer[(p.i_y_max * y) + x][0] = colors[color][0];
        image_buffer[(p.i_y_max * y) + x][1] = colors[color][1];
        image_buffer[(p.i_y_max * y) + x][2] = colors[color][2];
    };
};
*/
void write_to_file(Params p){
    FILE * file;
    const char * filename               = "output_cuda.ppm";
    const char * comment                = "# ";
    int max_color_component_value = 255;
    char* ch=(char*)malloc(3*sizeof(char));

    file = fopen(filename,"wb");
    
    fprintf(file, "P6\n %s\n %d\n %d\n %d\n", comment,
            p->i_x_max, p->i_y_max, max_color_component_value);
    printf("CCCCC\n");
    printf("%c\n",p->image_buffer[0]);
    printf("CCCCC\n");
    for(int i = 0; i < 3*p->image_buffer_size; i+=3){
        ch[0]=p->image_buffer[i];
        ch[1]=p->image_buffer[i+1];
        ch[2]=p->image_buffer[i+2];
        fwrite(ch, 1 , 3, file);
    };
    fclose(file);
    printf("CCCCC\n");
};

__global__ void compute_mandelbrot(Params p){


    double z_x;
    double z_y;
    double z_x_squared;
    double z_y_squared;
    double escape_radius_squared = 4;
    double c_x;
    double c_y;

    int iteration;
    uint i_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint i_y = (blockIdx.y * blockDim.y) + threadIdx.y;

    printf("%d,%d -",i_x,i_y);

    if(i_y<p->i_y_max){
        c_y = p->c_y_min + i_y * p->pixel_height;
        if(fabs(c_y) < p->pixel_height / 2){
            c_y = 0.0;
        };
        if(i_x<p->i_x_max){

            c_x         = p->c_x_min + i_x * p->pixel_width;

            z_x         = 0.0;
            z_y         = 0.0;

            z_x_squared = 0.0;
            z_y_squared = 0.0;

            for(iteration = 0;
                iteration < iteration_max && \
                ((z_x_squared + z_y_squared) < escape_radius_squared);
                iteration++){
                z_y         = 2 * z_x * z_y + c_y;
                z_x         = z_x_squared - z_y_squared + c_x;

                z_x_squared = z_x * z_x;
                z_y_squared = z_y * z_y;
            };

            int color;

            if(iteration == iteration_max){ 
                p->image_buffer[((p->i_y_max * i_y) + i_x)*3+0] = p->colors[gradient_size*3+0];
                p->image_buffer[((p->i_y_max * i_y) + i_x)*3+1] = p->colors[gradient_size*3+1];
                p->image_buffer[((p->i_y_max * i_y) + i_x)*3+2] = p->colors[gradient_size*3+2];

            }
            else{
                color = iteration % gradient_size;
        
                p->image_buffer[((p->i_y_max * i_y) + i_x)*3+0] = p->colors[color*3+0];
                p->image_buffer[((p->i_y_max * i_y) + i_x)*3+1] = p->colors[color*3+1];
                p->image_buffer[((p->i_y_max * i_y) + i_x)*3+2] = p->colors[color*3+2];
            };

        }
    }
};

int main(int argc, char *argv[]){
    Params p;
    p=CreateParams();
    timerAloc.c_start = clock();
    clock_gettime(CLOCK_MONOTONIC, &timerAloc.t_start);
    gettimeofday(&timerAloc.v_start, NULL);

    init(argc, argv,p);


    timer.c_start = clock();
    clock_gettime(CLOCK_MONOTONIC, &timer.t_start);
    gettimeofday(&timer.v_start, NULL);

    dim3 threadsPerBlock(8, 8); 
    dim3 numBlocks(p->image_size/8,p->image_size/8);

    compute_mandelbrot<<<numBlocks,threadsPerBlock>>>(p);    
    hipDeviceSynchronize();
    printf("BBBBBb\n");
    
    printf("%c ",p->image_buffer[0]);
    printf("BBBBBb\n");
    
    

    timer.c_end = clock();
    clock_gettime(CLOCK_MONOTONIC, &timer.t_end);
    gettimeofday(&timer.v_end, NULL);

    timerAloc.c_end = clock();
    clock_gettime(CLOCK_MONOTONIC, &timerAloc.t_end);
    gettimeofday(&timerAloc.v_end, NULL);

    write_to_file(p);
    printf("%f",
        (double) (timer.t_end.tv_sec - timer.t_start.tv_sec) +
        (double) (timer.t_end.tv_nsec - timer.t_start.tv_nsec) / 1000000000.0);
    
    printf (",");
    printf("%f",
        (double) (timerAloc.t_end.tv_sec - timerAloc.t_start.tv_sec) +
        (double) (timerAloc.t_end.tv_nsec - timerAloc.t_start.tv_nsec) / 1000000000.0);
    printf ("\n");

    hipFree(p->colors);
    hipFree(p->image_buffer);
    hipFree(p);

    return 0;
};
